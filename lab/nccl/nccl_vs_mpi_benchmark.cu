#include "hip/hip_runtime.h"
/*************************************************************************
 * NCCL vs MPI Communication Benchmark
 *
 * This application compares the performance of NCCL-based GPU communication
 * with traditional MPI-based Ethernet communication.
 ************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <mpi.h>
#include <math.h>

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                        \
    printf("CUDA error: %s\n", hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("NCCL error: %s\n", ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define MPICHECK(cmd) do {                          \
  int err = cmd;                                    \
  if (err != MPI_SUCCESS) {                         \
    char errstr[MPI_MAX_ERROR_STRING];              \
    int errlen;                                     \
    MPI_Error_string(err, errstr, &errlen);         \
    printf("MPI error: %s\n", errstr);              \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

typedef enum {
  OP_BROADCAST,
  OP_ALLREDUCE
} operation_t;

const char* operation_names[] = {
  "Broadcast",
  "AllReduce"
};

// Run NCCL operation and measure performance
double run_nccl_op(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                  ncclRedOp_t op, int root, int rank, ncclComm_t comm, hipStream_t stream, operation_t operation) {
  hipEvent_t start, stop;
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  // Warmup
  for (int i = 0; i < 5; i++) {
    if (operation == OP_BROADCAST) {
      NCCLCHECK(ncclBroadcast(sendbuff, recvbuff, count, datatype, root, comm, stream));
    } else if (operation == OP_ALLREDUCE) {
      NCCLCHECK(ncclAllReduce(sendbuff, recvbuff, count, datatype, op, comm, stream));
    }
  }
  CUDACHECK(hipStreamSynchronize(stream));

  // Benchmark
  int iterations = 20;
  CUDACHECK(hipEventRecord(start, stream));
  for (int i = 0; i < iterations; i++) {
    if (operation == OP_BROADCAST) {
      NCCLCHECK(ncclBroadcast(sendbuff, recvbuff, count, datatype, root, comm, stream));
    } else if (operation == OP_ALLREDUCE) {
      NCCLCHECK(ncclAllReduce(sendbuff, recvbuff, count, datatype, op, comm, stream));
    }
  }
  CUDACHECK(hipEventRecord(stop, stream));
  CUDACHECK(hipStreamSynchronize(stream));

  float milliseconds = 0;
  CUDACHECK(hipEventElapsedTime(&milliseconds, start, stop));
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));

  return milliseconds / iterations;
}

// Run MPI operation and measure performance
double run_mpi_op(void* sendbuff, void* recvbuff, size_t count, MPI_Datatype datatype,
                MPI_Op op, int root, int rank, int nranks, operation_t operation) {
  // Warmup
  for (int i = 0; i < 5; i++) {
    if (operation == OP_BROADCAST) {
      if (rank == root) {
        // For the root process, we need to use the sendbuff as source
        memcpy(recvbuff, sendbuff, count * sizeof(float));
      }
      MPICHECK(MPI_Bcast(recvbuff, count, datatype, root, MPI_COMM_WORLD));
    } else if (operation == OP_ALLREDUCE) {
      MPICHECK(MPI_Allreduce(sendbuff, recvbuff, count, datatype, op, MPI_COMM_WORLD));
    }
  }

  // Benchmark
  int iterations = 20;
  double start_time = MPI_Wtime();
  for (int i = 0; i < iterations; i++) {
    if (operation == OP_BROADCAST) {
      if (rank == root) {
        // For the root process, we need to use the sendbuff as source
        memcpy(recvbuff, sendbuff, count * sizeof(float));
      }
      MPICHECK(MPI_Bcast(recvbuff, count, datatype, root, MPI_COMM_WORLD));
    } else if (operation == OP_ALLREDUCE) {
      MPICHECK(MPI_Allreduce(sendbuff, recvbuff, count, datatype, op, MPI_COMM_WORLD));
    }
  }
  double end_time = MPI_Wtime();

  return ((end_time - start_time) * 1000.0) / iterations; // in milliseconds
}

// Initialize data for test
void init_data(float* data, size_t count, int rank) {
  for (size_t i = 0; i < count; i++) {
    data[i] = rank + 1.0f + (float)i / 1000.0f;
  }
}

// Verify the correctness of the data transfer
bool verify_data(float* result, float* expected, size_t count) {
  for (size_t i = 0; i < count; i++) {
    if (fabs(result[i] - expected[i]) > 1e-5) {
      printf("Data verification failed at index %zu: got %f, expected %f\n",
             i, result[i], expected[i]);
      return false;
    }
  }
  return true;
}

int main(int argc, char* argv[]) {
  // Initialize MPI
  int rank, nranks;
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nranks));

  // Get number of GPUs per node
  int device_count;
  CUDACHECK(hipGetDeviceCount(&device_count));

  // Select GPU based on local rank
  int local_rank = rank % device_count;
  CUDACHECK(hipSetDevice(local_rank));

  // Get device properties
  hipDeviceProp_t prop;
  CUDACHECK(hipGetDeviceProperties(&prop, local_rank));

  // Print system information
  // Have rank 0 print the header
  if (rank == 0) {
    printf("Running on %d processes\n", nranks);
  }

  // Barrier to ensure clean output
  MPI_Barrier(MPI_COMM_WORLD);

  // Each rank prints its GPU info
  printf("Rank %d using GPU %d: %s\n", rank, local_rank, prop.name);

  // Barrier before continuing
  MPI_Barrier(MPI_COMM_WORLD);

  // NCCL initialization
  ncclUniqueId nccl_id;
  ncclComm_t comm;
  hipStream_t stream;

  // Root process generates the NCCL ID and broadcasts it to all
  if (rank == 0) {
    NCCLCHECK(ncclGetUniqueId(&nccl_id));
  }
  MPICHECK(MPI_Bcast(&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD));

  // Initialize NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nranks, nccl_id, rank));
  CUDACHECK(hipStreamCreate(&stream));

  // Array of buffer sizes to test
  uint64_t buffer_sizes[] = {1ULL << 10, 1ULL << 14, 1ULL << 18, 1ULL << 20, 1ULL << 22, 1ULL << 24, 1ULL << 26, 1ULL << 28, 1ULL << 30, 1ULL << 32};
  int num_sizes = sizeof(buffer_sizes) / sizeof(buffer_sizes[0]);

  // Operations to test
  operation_t operations[] = {OP_BROADCAST, OP_ALLREDUCE};
  int num_operations = sizeof(operations) / sizeof(operations[0]);

  if (rank == 0) {
    printf("\n%20s | %15s | %15s | %15s | %15s | %10s\n",
           "Operation", "Size (bytes)", "NCCL Time (ms)", "MPI Time (ms)", "Speedup", "Verified");
    printf("----------------------------------------------------------------------------------------------------------\n");
  }

  // Main benchmark loop
  for (int op_idx = 0; op_idx < num_operations; op_idx++) {
    operation_t operation = operations[op_idx];
    int root = 0; // Root for broadcast operations

    for (int s = 0; s < num_sizes; s++) {
      size_t size = (size_t)buffer_sizes[s];
      size_t count = size / sizeof(float);

      // Allocate and initialize host data
      float *h_sendbuff, *h_recvbuff, *h_expected;
      h_sendbuff = (float*)malloc(size);
      h_recvbuff = (float*)malloc(size);
      h_expected = (float*)malloc(size);

      init_data(h_sendbuff, count, rank);
      memset(h_recvbuff, 0, size);

      // Prepare expected results for verification
      if (operation == OP_BROADCAST) {
        if (rank == root) {
          memcpy(h_expected, h_sendbuff, size);
        } else {
          float *root_data = (float*)malloc(size);
          init_data(root_data, count, root);
          memcpy(h_expected, root_data, size);
          free(root_data);
        }
      } else if (operation == OP_ALLREDUCE) {
        for (size_t i = 0; i < count; i++) {
          h_expected[i] = 0;
          for (int r = 0; r < nranks; r++) {
            h_expected[i] += r + 1.0f + (float)i / 1000.0f;
          }
        }
      }

      // Allocate device memory
      float *d_sendbuff, *d_recvbuff;
      CUDACHECK(hipMalloc(&d_sendbuff, size));
      CUDACHECK(hipMalloc(&d_recvbuff, size));

      // Copy data to device
      CUDACHECK(hipMemcpy(d_sendbuff, h_sendbuff, size, hipMemcpyHostToDevice));

      // For broadcast, root's receive buffer shouldn't be zero initialized
      if (operation == OP_BROADCAST && rank == root) {
        CUDACHECK(hipMemcpy(d_recvbuff, h_sendbuff, size, hipMemcpyHostToDevice));
      } else {
        CUDACHECK(hipMemset(d_recvbuff, 0, size));
      }

      // Run NCCL operation
      double nccl_time = run_nccl_op(
        d_sendbuff, d_recvbuff, count, ncclFloat,
        ncclSum, root, rank, comm, stream, operation
      );

      // Verify NCCL result
      CUDACHECK(hipMemcpy(h_recvbuff, d_recvbuff, size, hipMemcpyDeviceToHost));
      bool nccl_verified = verify_data(h_recvbuff, h_expected, count);

      // Clean up for NCCL test
      CUDACHECK(hipMemset(d_recvbuff, 0, size));

      // Run MPI operation
      memset(h_recvbuff, 0, size);
      double mpi_time = run_mpi_op(
        h_sendbuff, h_recvbuff, count, MPI_FLOAT,
        MPI_SUM, root, rank, nranks, operation
      );

      // Verify MPI result
      bool mpi_verified = verify_data(h_recvbuff, h_expected, count);

      // Calculate speedup
      double speedup = mpi_time / nccl_time;

      // Synchronize to ensure verification is complete
      MPI_Barrier(MPI_COMM_WORLD);

      // Aggregate verification results
      int nccl_verify_all = nccl_verified ? 1 : 0;
      int mpi_verify_all = mpi_verified ? 1 : 0;
      int nccl_verify_sum = 0;
      int mpi_verify_sum = 0;

      MPICHECK(MPI_Reduce(&nccl_verify_all, &nccl_verify_sum, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD));
      MPICHECK(MPI_Reduce(&mpi_verify_all, &mpi_verify_sum, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD));

      // Print results
      if (rank == 0) {
        bool all_verified = (nccl_verify_sum == nranks) && (mpi_verify_sum == nranks);
        printf("%20s | %15zu | %15.4f | %15.4f | %15.2fx | %10s\n",
               operation_names[operation], size, nccl_time, mpi_time, speedup,
               all_verified ? "Yes" : "No");
      }

      // Free memory
      CUDACHECK(hipFree(d_sendbuff));
      CUDACHECK(hipFree(d_recvbuff));
      free(h_sendbuff);
      free(h_recvbuff);
      free(h_expected);
    }
  }

  // Cleanup
  NCCLCHECK(ncclCommDestroy(comm));
  CUDACHECK(hipStreamDestroy(stream));
  MPICHECK(MPI_Finalize());

  return 0;
}